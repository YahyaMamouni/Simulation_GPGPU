#include "hip/hip_runtime.h"
#include <gpgpu.h>
#include <algorithm>
#include <iostream>
#include <random>
#include <assert.h>

//float4 FOX_COLOR = make_float4(1.0f, 0.5f, 0.0f, 1.0f);



// Override - 
__device__ float2 operator-(float2 a, float2 b) {
	return make_float2(a.x - b.x, a.y - b.y);
};

// random function

__device__ float fracf(float x)
{
	return x - floorf(x);
}

__device__ float random(float x, float y) {
	float t = 12.9898f * x + 78.233f * y;
	return abs(fracf(t * sin(t)));
}

void GetGPGPUInfo() {
	hipDeviceProp_t cuda_propeties;
	hipGetDeviceProperties(&cuda_propeties, 0);
	std::cout << "maxThreadsPerBlock: " << cuda_propeties.maxThreadsPerBlock << std::endl;
}

__global__ void kernel_uv(hipSurfaceObject_t surface, int32_t width, int32_t height, float time) {
	int32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	float u = (float)x / width;
	float v = (float)y / height;
	float4 color = make_float4(u, v, cos(time), 1.0f);
	surf2Dwrite(color, surface, x * sizeof(float4), y);
}

__global__ void kernel_copy(hipSurfaceObject_t surface_in, hipSurfaceObject_t surface_out) {
	int32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t y = blockIdx.y * blockDim.y + threadIdx.y;

	float4 color = make_float4(1.f, 0.f, 1.f, 1.0f);
	surf2Dread(&color, surface_in, x * sizeof(float4), y);
	surf2Dwrite(color, surface_out, x * sizeof(float4), y);
}


// Kernel thats draws the green background
__global__ void kernel_draw_map(hipSurfaceObject_t surface) {
	int32_t x = blockIdx.x * blockDim.x + threadIdx.x;
	int32_t y = blockIdx.y * blockDim.y + threadIdx.y;
	float4 color = make_float4(0.6f, 0.9f, 0.05f, 1.0f);

	surf2Dwrite(color, surface, x * sizeof(float4), y);
}


// Kernel to draw foxes
__global__ void DrawFoxes(hipSurfaceObject_t surface, Fox* fox_buffer, float4 fox_color, int width, int height) {
	// calculate the x and y coordinates for the current thread
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	float u = (float)x / width;
	float v = (float)y / height;

	// iterate through the buffer & draw circles that represent foxes
	for (int i = 0; i < MAX_FOX; i++) {
		if (fox_buffer[i].is_alive == true){
			if (hypotf(fox_buffer[i].u - u, fox_buffer[i].v - v) < fox_buffer[i].radius) {
				surf2Dwrite(fox_color, surface, sizeof(float4) * x, y, hipBoundaryModeTrap);
			}
		}
	}
}

// Kernel to draw rabbits
__global__ void DrawRabbits(hipSurfaceObject_t surface, Rabbit* rabbit_buffer, float4 rabbit_color, int width, int height) {
	// calculate the x and y coordinates for the current thread
	int x = threadIdx.x + blockIdx.x * blockDim.x;
	int y = threadIdx.y + blockIdx.y * blockDim.y;

	float u = (float)x / width;
	float v = (float)y / height;

	// iterate through the buffer & draw circles that represent rabbits
	for (int i = 0; i < MAX_RABBIT; i++) {
		if (rabbit_buffer[i].is_alive == true){
			if ((hypotf(rabbit_buffer[i].u - u, rabbit_buffer[i].v - v) < rabbit_buffer[i].radius)) {
				surf2Dwrite(rabbit_color, surface, sizeof(float4) * x, y, hipBoundaryModeTrap);
			}
		}
	}
}

// Kernel to move rabbits
__global__ void MoveRabbits(Rabbit* rabbit_buffer, int random_age, float random_u, float random_v) {
	int32_t index = threadIdx.x;
	if (rabbit_buffer[index].is_alive) {
		rabbit_buffer[index].age = rabbit_buffer[index].age + random_age;

		// calculating new dierctions
		float random_val = (random(rabbit_buffer[index].u, rabbit_buffer[index].v) - 0.5) * 3.14 / 4;
		float temp = rabbit_buffer[index].direction_u * cos(random_val) - rabbit_buffer[index].direction_v * sin(random_val);
		rabbit_buffer[index].direction_v = rabbit_buffer[index].direction_u * sin(random_val) + rabbit_buffer[index].direction_v * cos(random_val);
		rabbit_buffer[index].direction_u = temp;

		// In case we hit a border we inverse the direction
		if (rabbit_buffer[index].u >= 1 || rabbit_buffer[index].u <= 0) {
			rabbit_buffer[index].direction_u = 0.0f - rabbit_buffer[index].direction_u;
		}

		if (rabbit_buffer[index].v >= 1 || rabbit_buffer[index].v <= 0) {
			rabbit_buffer[index].direction_v = 0.0f - rabbit_buffer[index].direction_v;
		}
		// Moving rabbits
		rabbit_buffer[index].u = rabbit_buffer[index].u + (rabbit_buffer[index].direction_u / 1000);
		rabbit_buffer[index].v = rabbit_buffer[index].v + (rabbit_buffer[index].direction_v / 1000);
	}
}

// Kernel to move rabbits and spawn new ones
__global__ void SpawnRabbits(Rabbit* rabbit_buffer, int random_age, float random_u, float random_v) {
	int32_t index = threadIdx.x;
	// Add a random offset to the age threshold
	int age_threshold = random_age * 100;
	if (rabbit_buffer[index].age >= age_threshold) {
		for (int i = 0 ; i < MAX_RABBIT; i++) {
			if (rabbit_buffer[i].is_alive != true) {
				// Change previous position to avoid flickering
				rabbit_buffer[i].u = random_u;
				rabbit_buffer[i].v = random_v;
				rabbit_buffer[i].is_alive = true;
				rabbit_buffer[i].age = 0;
				break;
			}
		}
	}

}

// Kernel to move foxes
__global__ void MoveFoxes(Fox* fox_buffer, int random_age) {
	int32_t index = threadIdx.x;
	if (fox_buffer[index].is_alive) {
		fox_buffer[index].age = fox_buffer[index].age + random_age;

		// calculating new dierctions
		float random_val = (random(fox_buffer[index].u, fox_buffer[index].v) - 0.5) * 3.14 / 4;
		float temp = fox_buffer[index].direction_u * cos(random_val) - fox_buffer[index].direction_v * sin(random_val);
		fox_buffer[index].direction_v = fox_buffer[index].direction_u * sin(random_val) + fox_buffer[index].direction_v * cos(random_val);
		fox_buffer[index].direction_u = temp;


		// In case we hit a border we inverse the direction
		if (fox_buffer[index].u >= 1 || fox_buffer[index].u <= 0) {
			fox_buffer[index].direction_u = 0.0f - fox_buffer[index].direction_u;
		}
		if (fox_buffer[index].v >= 1 || fox_buffer[index].v <= 0) {
			fox_buffer[index].direction_v = 0.0f - fox_buffer[index].direction_v;
		}
		// Moving foxes
		fox_buffer[index].u = fox_buffer[index].u + (fox_buffer[index].direction_u / 1000);
		fox_buffer[index].v = fox_buffer[index].v + (fox_buffer[index].direction_v / 1000);
	}
}

// Kernel to spawn foxes
__global__ void SpawnFoxes(Fox* fox_buffer, int random_age, float random_u, float random_v) {
	int32_t index = threadIdx.x;
	// Add a random offset to the age threshold
	int age_threshold = random_age * 50;
	if (fox_buffer[index].age >= age_threshold) {
		for (int i = 0 ; i < MAX_FOX; i++) {
			if (fox_buffer[i].is_alive != true) {
				// Change previous position to avoid flickering
				fox_buffer[i].u = random_u;
				fox_buffer[i].v = random_v;
				fox_buffer[i].is_alive = true;
				fox_buffer[i].age = 0;
				break;
			}
		}
	}

}


// Kill rabbits (Not atomic)
/*
__global__ void KillRabbits(hipSurfaceObject_t surface, Rabbit* rabbit_buffer, Fox* fox_buffer) {
	int32_t index = threadIdx.x;

	for (int i = 0; i < MAX_FOX; i++) {

		if (fox_buffer[i].is_alive == true && rabbit_buffer[i].is_alive == true){
			if (hypotf(fox_buffer[i].u - rabbit_buffer[index].u, fox_buffer[i].v - rabbit_buffer[index].v) < fox_buffer[i].radius + 20) {
				rabbit_buffer[index].is_alive = false;
			}
			else{
				fox_buffer[i].death--;
			}
		}
	}

}*/

// Not atomic
/*
__global__ void ChaseRabbitsAndKillFoxes(hipSurfaceObject_t surface, Fox* fox_buffer) {
	int32_t index = threadIdx.x;

		if (fox_buffer[index].is_alive == true && fox_buffer[index].death == 0){
			fox_buffer[index].is_alive = false;
		}
}*/


// Kernel to chase rabbits
__global__ void ChaseRabbits(Fox* fox_buffer, Rabbit* rabbit_buffer) {
	int32_t index = threadIdx.x;

	// Chase
	for (int i = 0; i < MAX_RABBIT; i++) {

		if (rabbit_buffer[i].is_alive == true && hypotf(fox_buffer[index].u - rabbit_buffer[i].u, fox_buffer[index].v - rabbit_buffer[i].v) < fox_buffer[index].radius + 0.03) {
			fox_buffer[index].direction_u = rabbit_buffer[i].u - fox_buffer[index].u;
			fox_buffer[index].direction_v = rabbit_buffer[i].v - fox_buffer[index].v;
			float norm = sqrt((fox_buffer[index].direction_u * fox_buffer[index].direction_u) + (fox_buffer[index].direction_v * fox_buffer[index].direction_v));
			fox_buffer[index].direction_u = fox_buffer[index].direction_u / norm;
			fox_buffer[index].direction_v = fox_buffer[index].direction_v / norm;
		}
	}
}

// Kernel to chase rabbits and kill foxes if they acheive a certain age
__global__ void KillFoxes(Fox* fox_buffer) {
	int32_t index = threadIdx.x;

	if (fox_buffer[index].is_alive == true && fox_buffer[index].death == 0) {
		atomicExch(&fox_buffer[index].is_alive, 0);
	}
}


// (Atomic) Kernel where foxes kill rabbits if they are in a certain radius
__global__ void KillRabbits(hipSurfaceObject_t surface, Rabbit* rabbit_buffer, Fox* fox_buffer) {
    int32_t index = threadIdx.x;

    for (int i = 0; i < MAX_FOX; i++) {
        if (fox_buffer[i].is_alive == true && rabbit_buffer[i].is_alive == true){
            if (hypotf(fox_buffer[i].u - rabbit_buffer[index].u, fox_buffer[i].v - rabbit_buffer[index].v) < fox_buffer[i].radius + 0.005) {
				atomicExch(&rabbit_buffer[index].is_alive, 0);      
            }
			else {
				fox_buffer[i].death--;
			}
        }
    }
}



void DrawUVs(hipSurfaceObject_t surface, int32_t width, int32_t height, float time) {
	dim3 threads(32, 32);
	dim3 blocks(32, 32);
	kernel_uv << <blocks, threads >> > (surface, width, height, time);
}

void CopyTo(hipSurfaceObject_t surface_in, hipSurfaceObject_t surface_out, int32_t width, int32_t height) {
	dim3 threads(32, 32);
	dim3 blocks(32, 32);
	kernel_copy << <blocks, threads >>> (surface_in, surface_out);
}

void destroy(Fox* device_foxes, Rabbit* device_rabbits) {
	// free device-side memory
	hipFree(device_foxes);
	hipFree(device_rabbits);
}


// Function that calls all the used kernels
void DrawMap(hipSurfaceObject_t surface, int32_t width, int32_t height, float time, int number_foxes, int * number_rabbits) {
	dim3 threads(32, 32);
	dim3 blocks(32, 32);

	// Animal colors
	float4 fox_color = make_float4(1.0f, 0.5f, 0.0f, 1.0f);
	float4 rabbit_color = make_float4(1.0f, 1.0f, 1.0f, 1.0f);

	// Draw the green background
	kernel_draw_map << <blocks, threads >> > (surface);

	// Bool used so we can init data only once in the beginning
	static bool is_init = false;

	// Device buffers
	static Fox* device_foxes;
	static Rabbit* device_rabbits;
	// Host buffers
	Fox* fox_buffer = new Fox[MAX_FOX];
	Rabbit* rabbit_buffer = new Rabbit[MAX_RABBIT];

	// Generate random u & v
	std::random_device rd;
	std::mt19937 gen(rd());
	std::uniform_real_distribution<> x(0.0, 1.0);
	std::uniform_real_distribution<> y(0.0, 1.0);

	// Init only once then jump always
	if (!is_init) {
		is_init = true;

		// random number to generate random directions
		std::uniform_real_distribution<> direction_x(-1.0, 1.0);
		
		// Init foxes
		for (int i = 0; i < MAX_FOX; i++) {
			fox_buffer[i].u = x(gen);
			fox_buffer[i].v = y(gen);
			float dir_u = direction_x(gen);
			float dir_v = direction_x(gen);
			float norme = sqrt((dir_u * dir_u) + (dir_v * dir_v));
			fox_buffer[i].direction_u = dir_u / norme;
			fox_buffer[i].direction_v = dir_v / norme;
			fox_buffer[i].radius = 0.008;
			if (i < number_foxes){
				fox_buffer[i].is_alive = true;
			}
		}

		// Init rabbits
		for (int i = 0; i < MAX_RABBIT; i++) {
			rabbit_buffer[i].u = x(gen);
			rabbit_buffer[i].v = y(gen);
			float dir_u = direction_x(gen);
			float dir_v = direction_x(gen);
			float norme = sqrt((dir_u * dir_u) + (dir_v * dir_v));
			rabbit_buffer[i].direction_u = dir_u / norme;
			rabbit_buffer[i].direction_v = dir_v / norme;
			rabbit_buffer[i].radius = 0.007;
			if (i < *number_rabbits){
				rabbit_buffer[i].is_alive = true;
			}
		}

		// Allocate device-side memory
		
		hipMalloc(&device_foxes, sizeof(Fox) * MAX_FOX);

		hipMalloc(&device_rabbits, sizeof(Rabbit) * MAX_RABBIT);

		// copy data to device
		hipMemcpy(device_foxes, fox_buffer, sizeof(Fox) * MAX_FOX, hipMemcpyHostToDevice);
		hipMemcpy(device_rabbits, rabbit_buffer, sizeof(Rabbit) * MAX_RABBIT, hipMemcpyHostToDevice);
	}


	DrawFoxes << <blocks, threads >> > (surface, device_foxes, fox_color, width, height);
	DrawRabbits << <blocks, threads >> > (surface, device_rabbits, rabbit_color, width, height);
	assert(*number_rabbits <= 500);
	std::uniform_real_distribution<> random_age(0, 10);
	MoveRabbits <<<1, MAX_RABBIT >>> (device_rabbits, random_age(gen), x(gen), y(gen));
	SpawnRabbits << <1, MAX_RABBIT >> > (device_rabbits, random_age(gen), x(gen), y(gen));
	MoveFoxes << <1, MAX_FOX >> > (device_foxes, random_age(gen));
	SpawnFoxes << <1, MAX_FOX >> > (device_foxes, random_age(gen), x(gen), y(gen));
	ChaseRabbits <<<1, MAX_FOX >>> (device_foxes, device_rabbits);
	KillFoxes << <1, MAX_FOX >> > (device_foxes);
	KillRabbits << <1, MAX_RABBIT >> > (surface, device_rabbits, device_foxes);
}
